#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <hip/hip_runtime.h>
#include <chrono>
#include <sstream>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>

#define CUDA_CHECK(call)                                                                                                  \
    {                                                                                                                     \
        hipError_t err = call;                                                                                           \
        if (err != hipSuccess)                                                                                           \
        {                                                                                                                 \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__ << ": " << hipGetErrorString(err) << std::endl; \
            exit(EXIT_FAILURE);                                                                                           \
        }                                                                                                                 \
    }

#define BitsInInt 32

// Srructure representing a node in the trie
struct Node {
    int *bit;  // bit value of the node
    int *bit0; // index of the child node with bit value 0
    int *bit1; // index of the child node with bit value 1
};


// Function to load the trie and the sequence vector from a file
void load_trie(const char *filename, std::vector<int> &bit, std::vector<int> &bit0, std::vector<int> &bit1, std::vector<int> &sequences, int &sequence_length, int &num_sequences) {
    std::ifstream file(filename);
    if (!file.is_open()) {
        std::cerr << "Error: could not open file " << filename << std::endl;
        exit(EXIT_FAILURE);
    }

    // Add the root node which bit value does not matter
    bit.push_back(-1);
    bit0.push_back(-1);
    bit1.push_back(-1);


    // the first line represents sequence length and number of sequences
    std::string line;
    if (!std::getline(file, line)) {
        std::cerr << "Error: could not read trie" << std::endl;
        exit(EXIT_FAILURE);
    }
    std::istringstream iss(line);
    if (!(iss >> sequence_length >> num_sequences)) {
        std::cerr << "Error: could not parse sequence length and number of sequences" << std::endl;
        exit(EXIT_FAILURE);
    }

    // Seqences will be represented as 32 bit integers
    // num_ints is the number of integers needed to represent the sequence
    int num_ints = (sequence_length + BitsInInt - 1) / BitsInInt;

    // padding_zeros is the number of zeros needed to fill the last integer
    int padding_zeros = (num_ints * BitsInInt) - sequence_length;


    // Read the sequences
    for (int i = 0; i < num_sequences; i++) {
        std::string sequence;
        if (!(file >> sequence)) {
            std::cerr << "Error: could not read sequence " << i << std::endl;
            break;
        }


        if (sequence.length() != static_cast<std::string::size_type>(sequence_length)) {
            std::cerr << "Error: invalid sequence length (" << sequence.length()
                      << ") for sequence " << i << std::endl;
            continue;
        }

        // Convert the sequence to integers and store them in the sequences vector
        for (int j = 0; j < num_ints; j++) {
            int chunk = 0;
            int valid_bits = BitsInInt;

            if (j == num_ints - 1) {
                valid_bits = BitsInInt - padding_zeros;
            }

            for (int k = 0; k < valid_bits; k++) {
                char bit_char = sequence[j * BitsInInt + k];
                int bit = (bit_char == '1') ? 1 : 0;
                chunk |= bit << (valid_bits - 1 - k);
            }
            // Add the chunk to the sequences vector
            sequences.push_back(chunk);
        }

        int current = 0;

        // Build the trie
        for (int bit_pos = 0; bit_pos < sequence_length; ++bit_pos) {

            // Get the bit value at the current bit position
            char bit_char = sequence[bit_pos];
            int current_bit = (bit_char == '1') ? 1 : 0;

            // Get the index of the child node with the current bit
            int childIdx = (current_bit == 0) ? bit0[current] : bit1[current];

            // If the child does not exist, create it
            if (childIdx == -1) {
                childIdx = bit.size();
                bit.push_back(current_bit);
                bit0.push_back(-1);
                bit1.push_back(-1);

                if (current_bit == 0) {
                    bit0[current] = childIdx;
                } else {
                    bit1[current] = childIdx;
                }
            }

            current = childIdx;            
        }

    }
    file.close();
}

// Helper function to print the trie
void print_tire(std::vector<int> &bit, std::vector<int> &bit0, std::vector<int> &bit1) {
    for (size_t i = 0; i < bit.size(); i++) {
        std::cout << "Bit " << bit[i] << std::endl;
        std::cout << "  Bit0 " << bit0[i] << std::endl;
        std::cout << "  Bit1 " << bit1[i] << std::endl;
    }
}

// Kernel to find the pairs
__global__ void find_pairs_kernel(int *d_sequences, const int *d_bit, const int *d_bit0, const int *d_bit1, int l, int n, int *d_sequence_idx, int *d_flipped_bit_position, int *d_pairs_found)
{
    // Get the sequence index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // If the sequence index is out of bounds, return
    if (idx >= n)
        return;

    // Number of integers needed to represent the sequence and the number of zeros needed to fill the last integer
    int num_ints = (l + BitsInInt - 1) / BitsInInt;
    int padding_zeros = (num_ints * BitsInInt) - l;
    
    // Get the sequence
    int *seq = d_sequences + idx * num_ints;

    // Iterate over the sequence chunks
    for (int chunk_idx = 0; chunk_idx < num_ints; ++chunk_idx) {
        // Number of valid bits in the chunk regarding the padding zeros
        int valid_bits = BitsInInt;

        // If it is the last chunk, calculate the number of valid bits
        if (chunk_idx == num_ints - 1) {
            valid_bits = BitsInInt - padding_zeros;
        }

        // Get the chunk
        int chunk = seq[chunk_idx];

        // Iterate over the bits in the chunk flipping them one by one and checking if the sequence exists in the trie
        for (int bit_pos = 0; bit_pos < valid_bits; ++bit_pos) {
            // Flip the bit at the current position
            unsigned int flipped_chunk = chunk ^ (1 << (valid_bits - 1 - bit_pos));

            // Flag to check if all chunks exist
            bool all_chunks_exist = true;
            
            int current = 0;

            // Iterate over the chunks and check if they exist in the trie
            for (int check_chunk_idx = 0; check_chunk_idx < num_ints; ++check_chunk_idx) {
                unsigned int check_chunk = 0;
                int check_bits = BitsInInt;

                if(check_chunk_idx == num_ints - 1) {
                    check_bits = BitsInInt - padding_zeros;
                }

                // Get the chunk
                if (check_chunk_idx == chunk_idx) {
                    check_chunk = flipped_chunk;
                } else {
                    check_chunk = seq[check_chunk_idx];
                }

                // Traverse the trie
                for (int k = 0; k < check_bits; ++k) {
                    // Get the bit at the current position and move to the child node
                    int bit = (check_chunk >> (check_bits - 1 - k)) & 1;
                    current = (bit == 0) ? d_bit0[current] : d_bit1[current];
                    
                    // If the child does not exist, break
                    if (current == -1) {
                        all_chunks_exist = false;
                        break;
                    }
                }

                if (!all_chunks_exist)
                    break;
            }

            // If the sequence exists, add it to the results
            if (all_chunks_exist) {
                int pair_idx = atomicAdd(d_pairs_found, 1);
                d_sequence_idx[pair_idx] = idx;
                d_flipped_bit_position[pair_idx] = chunk_idx * BitsInInt + bit_pos;
            }
        }
    }
}


int main() {
    // Host structure of arrays
    std::vector<int> h_bit;
    std::vector<int> h_bit0;
    std::vector<int> h_bit1;
    std::vector<int> h_sequences;
    int sequence_length;
    int num_sequences;

    auto start_time_load = std::chrono::high_resolution_clock::now();
    load_trie("input_sequences.txt", h_bit, h_bit0, h_bit1, h_sequences, sequence_length, num_sequences);
    auto end_time_load = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> load_duration = end_time_load - start_time_load;
    printf("GPU: Time to load trie and sequences: %f seconds\n", load_duration.count());
    size_t trie_size = h_bit.size();

    // Device structure of arrays
    auto start_time_alloc = std::chrono::high_resolution_clock::now();
    Node d_trie;
    CUDA_CHECK(hipMalloc(&d_trie.bit, trie_size * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_trie.bit0, trie_size * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_trie.bit1, trie_size * sizeof(int)));

    CUDA_CHECK(hipMemcpy(d_trie.bit, h_bit.data(), trie_size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_trie.bit0, h_bit0.data(), trie_size * sizeof(int), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_trie.bit1, h_bit1.data(), trie_size * sizeof(int), hipMemcpyHostToDevice));

    // Sequence memory
    int *d_sequences;
    CUDA_CHECK(hipMalloc(&d_sequences, h_sequences.size() * sizeof(int)));
    CUDA_CHECK(hipMemcpy(d_sequences, h_sequences.data(), h_sequences.size() * sizeof(int), hipMemcpyHostToDevice));

    // Device memory for storing the results
    int *d_pairs_found;
    int *d_sequence_idx;
    int *d_flipped_bit_position;
    CUDA_CHECK(hipMalloc(&d_pairs_found, sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_sequence_idx, sequence_length * num_sequences * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_flipped_bit_position, sequence_length * num_sequences * sizeof(int)));
    CUDA_CHECK(hipMemset(d_pairs_found, 0, sizeof(int)));

    auto end_time_alloc = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> alloc_duration = end_time_alloc - start_time_alloc;
    printf("GPU: Time to allocate memory and copy to device: %f seconds\n", alloc_duration.count());
    
    // Launch kernel
    int block_size = 1024;
    int grid_size = (num_sequences + block_size - 1) / block_size;

    auto start_time = std::chrono::high_resolution_clock::now();
    find_pairs_kernel<<<grid_size, block_size>>>(d_sequences, d_trie.bit, d_trie.bit0, d_trie.bit1, sequence_length, num_sequences, d_sequence_idx, d_flipped_bit_position, d_pairs_found);
    CUDA_CHECK(hipDeviceSynchronize());
    auto end_time = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> find_duration = end_time - start_time;
    printf("GPU: Time to find pairs: %f seconds\n", find_duration.count());

    // Copy results back to host
    auto start_time_copy = std::chrono::high_resolution_clock::now();
    int pairs_found;
    CUDA_CHECK(hipMemcpy(&pairs_found, d_pairs_found, sizeof(int), hipMemcpyDeviceToHost));
    std::vector<int> h_sequence_idx(pairs_found);
    std::vector<int> h_flipped_bit_position(pairs_found);

    CUDA_CHECK(hipMemcpy(h_sequence_idx.data(), d_sequence_idx, pairs_found * sizeof(int), hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(h_flipped_bit_position.data(), d_flipped_bit_position, pairs_found * sizeof(int), hipMemcpyDeviceToHost));

    auto end_time_copy = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> copy_duration = end_time_copy - start_time_copy;
    printf("GPU: Time to copy results to host: %f seconds\n", copy_duration.count());
    
    // Print results
    std::ofstream outfile("gpu_output_pairs.txt");
    for (int i = 0; i < pairs_found; i++) {
       outfile << "Sequence " << h_sequence_idx[i] << " flipped at bit " << h_flipped_bit_position[i] << std::endl;
    }
    outfile.close();



    // Cleanup
    CUDA_CHECK(hipFree(d_trie.bit));
    CUDA_CHECK(hipFree(d_trie.bit0));
    CUDA_CHECK(hipFree(d_trie.bit1));
    CUDA_CHECK(hipFree(d_sequences));
    CUDA_CHECK(hipFree(d_pairs_found));
    CUDA_CHECK(hipFree(d_sequence_idx));
    CUDA_CHECK(hipFree(d_flipped_bit_position));

    return 0;
}